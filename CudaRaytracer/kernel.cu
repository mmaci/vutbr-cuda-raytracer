#include "hip/hip_runtime.h"
#include <iostream>
#include "constants.h"

#include "ray.h"
#include "sphere.h"
#include "mathematics.h"
#include "camera.h"
#include "plane.h"
#include "light.h"
#include "scene.h"
#include "phong.h"

using namespace CUDA;


/**
* Checks for error and if found writes to cerr and exits program. 
*/
void checkCUDAError()
{
	hipError_t err = hipGetLastError();
	if(hipSuccess != err)
	{
		std::cerr << "Cuda error: " << hipGetErrorString(err) << std::endl;
		exit(EXIT_FAILURE);
	}
}


__device__ HitInfo Intersect(const Ray &ray,  Plane* p,  Sphere* s, SceneStats* sceneStats){

	HitInfo hitInfo;
	int i;
	float st,pt;
	st = 0.f;
	pt = 0.f;
	int maxPi,maxSi;
	float tmp;
	for (i=0;i<(sceneStats->SphereCount);i++){
		tmp = s[i].intersect(ray);
		if (st < tmp){
			st = tmp;
			maxSi =i;
		}				
	}
	for (i=0;i<(sceneStats->PlaneCount);i++){
		tmp = p[i].intersect(ray);
		if (pt < tmp){
			pt = tmp;
			maxPi = i;
		}				
	}
	if ((pt==0.f) && (st == 0.f))//miss
	{ 
		//t = 0.f;
		hitInfo.hit = false;
		return hitInfo;
		//return false;
	} else if (pt > st) //plane hit
	{
		hitInfo.point = ray.getPoint(pt);
		hitInfo.normal = p[maxPi].normal;
		//t = pt;
		hitInfo.phongInfo = p[maxPi].phong;
	}else if (st >= pt) //sphere hit
	{
		hitInfo.point = ray.getPoint(st);
		hitInfo.normal = s[maxSi].getNormal(hitInfo.point);
		//t = st;
		hitInfo.phongInfo = s[maxSi].phong;

	}
	hitInfo.hit = true;
	return hitInfo;
	//return true;
}

__device__ Color TraceRay(const Ray &ray,  Plane* p,  Sphere* s, PointLight &light, SceneStats* sceneStats, int recursion)
{
	Color color(0,0,0);


	HitInfo hitInfo=Intersect(ray,p,s,sceneStats);
	if (hitInfo.hit){
		color = hitInfo.phongInfo.ambient;
		//light
		Ray sray = light.getShadowRay(hitInfo.point);
		sray.ShiftStart();
		HitInfo sHit = Intersect(sray,p,s,sceneStats);
		if (sHit.hit) {
			float3 lv = sray.direction;
			lv = CUDA::normalize(lv);
			color.accumulate(mult(hitInfo.phongInfo.diffuse,light.color),fabs(CUDA::dot(lv,hitInfo.normal)));
			if (hitInfo.phongInfo.shininess != 0) {
				float3 rlv = float3_sub(
					CUDA::cross(
					float3_mult(2.f,
					CUDA::cross(lv,hitInfo.normal)), hitInfo.normal) , lv);
				float3 vv = ray.direction;
				vv = CUDA::normalize(vv);
				float specular = 0.f-CUDA::dot(rlv,vv);
				if (specular > 0) {
					color.accumulate(mult(hitInfo.phongInfo.specular, light.color), pow(specular, hitInfo.phongInfo.shininess));
				}
			}

		}
		//reflected ray
		if (hitInfo.phongInfo.reflectance && recursion > 0) {
			Ray rray(hitInfo.point, float3_sub(ray.direction, CUDA::cross(float3_mult(2,CUDA::cross(ray.direction,hitInfo.normal)) ,hitInfo.normal)));
			rray.ShiftStart(1e-5);
			//Color rcolor = TraceRay(rray, p,s, light,sceneStats, recursion-1);
			//        color *= 1-phong.GetReflectance();
			//color.accumulate(rcolor, hitInfo.phongInfo.reflectance);
		}
		return color;
	}else
	{
		return Color(0,0,0);
	}

}	

/**
* CUDA kernel
*
* @param uchar* data
* @param uint32 width
* @param uint32 height
* @param float time
*/
__global__ void RTKernel(uchar4* data, uint32 width, uint32 height, Sphere* spheres, Plane* planes, SceneStats* sceneStats, Camera* camera)
{
	uint32 X = (blockIdx.x * blockDim.x) + threadIdx.x;
	uint32 Y = (blockIdx.y * blockDim.y) + threadIdx.y;

	float x = (2.f*X/WINDOW_WIDTH - 1.f);
	float y = (2.f*Y/WINDOW_HEIGHT - 1.f);


	Ray ray = camera->getRay(x,y);

	PointLight l(make_float3(-4.f, -5.f, 2.f),Color(0.5,0.5,0.5));
	//Color c = TraceRay(ray,scene,l,15);
	Color c = TraceRay(ray, planes, spheres, l,sceneStats, 15);
	data[WINDOW_WIDTH * Y + X].x = min(c.red*255.f,255.f);
	data[WINDOW_WIDTH * Y + X].y = min(c.green*255.f,255.f);
	data[WINDOW_WIDTH * Y + X].z = min(c.blue*255.f,255.f);
	data[WINDOW_WIDTH * Y + X].w = 0;
	/*data[WINDOW_WIDTH * Y + X].x = c.red*255.f;
	data[WINDOW_WIDTH * Y + X].y = c.green*255.f ;
	data[WINDOW_WIDTH * Y + X].z = c.blue*255.f ;
	data[WINDOW_WIDTH * Y + X].w = 0;*/

}


/**
* Wrapper for the CUDA kernel
*
* @param uchar* data
* @param uint32 width
* @param uint32 height
* @param float time
*/
extern "C" void launchRTKernel(uchar4* data, uint32 imageWidth, uint32 imageHeight, Sphere* spheres, Plane* planes, SceneStats* sceneStats, Camera* camera)
{   	
	dim3 threadsPerBlock(8, 8, 1); // 64 threads ~ 8*8
	dim3 numBlocks(WINDOW_WIDTH / threadsPerBlock.x, WINDOW_HEIGHT / threadsPerBlock.y);

	RTKernel<<<numBlocks, threadsPerBlock>>>(data, imageWidth, imageHeight, spheres, planes, sceneStats, camera);

	hipDeviceSynchronize();
	checkCUDAError();
}
