#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include "contants.h"

#include <GL/glew.h>
#include <glm.hpp>


#define WINDOW 800

GLuint vbo;

__global__ 
void hello(char *a, int *b) 
{
	a[threadIdx.x] += b[threadIdx.x];
}
 
int main()
{
	glBindBuffer(GL_ARRAY_BUFFER, vbo);
	glVertexPointer(2, GL_FLOAT, 12, 0);
	glColorPointer(4,GL_UNSIGNED_BYTE,12,(GLvoid*)8);

	glEnableClientState(GL_VERTEX_ARRAY);
	glEnableClientState(GL_COLOR_ARRAY);
	glDrawArrays(GL_POINTS, 0, WINDOW * WINDOW);
	glDisableClientState(GL_VERTEX_ARRAY);

	return EXIT_SUCCESS;
}
