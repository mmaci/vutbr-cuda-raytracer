#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include "contants.h"

__global__ 
void hello(char *a, int *b) 
{
	a[threadIdx.x] += b[threadIdx.x];
}
 
int main()
{
	char a[N] = "Hello \0\0\0\0\0\0";
	int b[N] = {15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
 
	char *ad;
	int *bd;
	const int csize = N*sizeof(char);
	const int isize = N*sizeof(int);
 
	printf("%s", a);
 
	hipMalloc( (void**)&ad, csize ); 
	hipMalloc( (void**)&bd, isize ); 
	hipMemcpy( ad, a, csize, hipMemcpyHostToDevice ); 
	hipMemcpy( bd, b, isize, hipMemcpyHostToDevice ); 
	
	dim3 dimBlock( blocksize, 1 );
	dim3 dimGrid( 1, 1 );
	hello<<<dimGrid, dimBlock>>>(ad, bd);
	hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost ); 
	hipFree( ad );
	hipFree( bd );
	
	printf("%s\n", a);
	return EXIT_SUCCESS;
}
